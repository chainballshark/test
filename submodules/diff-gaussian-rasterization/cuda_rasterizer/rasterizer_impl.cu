#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"


hipEvent_t start, stop;




__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	return glm::max(result, 0.0f);
}


// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)//二分法查找最高位比特位
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.




__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

__global__ void compute_fov_colors(
	const int P, const int D, const int M,
	const float* orig_points,
	const glm::vec3* cam_pos,
	const float* shs,
	const int* __restrict__ radii,
	float* __restrict__ rgb)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	if (radii[idx] > 0)
	{
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs);
		rgb[idx * 3 + 0] = result.x;
		rgb[idx * 3 + 1] = result.y;
		rgb[idx * 3 + 2] = result.z;
	}
}

// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void OBB_test(
	int P,
	const float2* points_xy,
	const uint32_t* offsets,
	const dim3 grid,
	int* radii,
	float* eigen_lengths,
	float* eigen_vecs,
	uint32_t* tiles_touched,
	bool* OBB_result)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;
		float2 point_image = points_xy[idx];
		getRect(point_image, radii[idx], rect_min, rect_max, grid);

        int potential_tnum = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);

		int count=0;

		
		if (potential_tnum == 1)
		{
			count = 1;
		}
		else
		{
			float2 eigenvector1 = { eigen_vecs[idx * 4], eigen_vecs[idx * 4 + 1] };
			float2 eigenvector2 = { eigen_vecs[idx * 4 + 2], eigen_vecs[idx * 4 + 3] };
			float len1 = eigen_lengths[idx * 2];
			float len2 = eigen_lengths[idx * 2 + 1];
			float2 center = point_image;

			// get 4 vertex from center and len1, len2, eigenvector1, eigenvector2
			float d1x = len1 * eigenvector1.x;
			float d1y = len1 * eigenvector1.y;
			float d2x = len2 * eigenvector2.x;
			float d2y = len2 * eigenvector2.y;
			float2 vertexs[4] = { 
				{center.x + d1x + d2x , center.y + d1y + d2y},
				{center.x - d1x + d2x , center.y - d1y + d2y},
				{center.x - d1x - d2x , center.y - d1y - d2y},
				{center.x + d1x - d2x , center.y + d1y - d2y}
			};



			for (int y = rect_min.y; y < rect_max.y; y++)
			{
				for (int x = rect_min.x; x < rect_max.x; x++)
				{
					float px = float(x) * float(BLOCK_X) +  float(BLOCK_X) / 2.0f;
					float py = float(y) * float(BLOCK_Y) + float(BLOCK_Y) / 2.0f;
					bool inside;
					//OBB_check(px, py, vertexs, center, eigenvector1, eigenvector2, len1, len2, inside);
	                OBB_check_SAT(px,py, vertexs, center,eigenvector1, eigenvector2, inside);
                    printf("inside:%d\n",inside);
			    	if (inside)
			  		{
						count++;
					}
					OBB_result[off] = inside;
					off++;
				}
			}
		}
		tiles_touched[idx] = count;
		if (count == 0)
		{
			radii[idx] = 0;
		}
	}
}
// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(//根据高斯点的位置和深度信息生成键值对，用于后续的排序或其他操作。
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets_old,
	const uint32_t* offsets,
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,//高斯点的值
	int* radii,
	dim3 grid,
	bool* OBB_result)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint32_t off_old = (idx == 0) ? 0 : offsets_old[idx - 1];
		uint2 rect_min, rect_max;

		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		int potential_tnum = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);

		if (potential_tnum == 1)//单个tile
		{
			uint64_t key = rect_min.y * grid.x + rect_min.x;
			key <<= 32;
			key |= *((uint32_t*)&depths[idx]);
			gaussian_keys_unsorted[off] = key;
			gaussian_values_unsorted[off] = idx;
			off++;
		}
		else//多个tile
		{
			for (int y = rect_min.y; y < rect_max.y; y++)
			{
				for (int x = rect_min.x; x < rect_max.x; x++)
				{
					bool inside = OBB_result[off_old];
					off_old++;
					if (inside)
					{
						uint64_t key = y * grid.x + x;
						key <<= 32;
						key |= *((uint32_t*)&depths[idx]);
						gaussian_keys_unsorted[off] = key;
						gaussian_values_unsorted[off] = idx;
						off++;
					}
				}
			}
		}
    }

}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)//瓦片的起始和结束索引
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];//当前高斯索引的列表
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;//右移32位，提取tile的id

		if (currtile != prevtile)//将当前线程的前一个线程的瓦片 ID 提取出来，并与当前线程的瓦片 ID 进行比较。
		{ // // check if prevtile is in the range
			// if (currtile > num_tiles - 1 || prevtile > num_tiles - 1|| currtile < 0 || prevtile < 0)
			// {
			// 	printf("Error: currtile: %d, prevtile: %d, num_tiles: %d\n", currtile, prevtile, num_tiles);
			// } 
			ranges[prevtile].y = idx;//如果两者不相同，则说明进入了新的瓦片，需要更新前一个瓦片的结束索引和当前瓦片的起始索引。
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	std::function<char* (size_t)> OBBFunc,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color,
	int* radii,
    float* eigen_lengths,	
	float* eigen_vecs,
	bool debug)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	size_t chunk_size = required<GeometryState>(P);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	size_t img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}
      /* hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); */
	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	CHECK_CUDA(FORWARD::preprocess(//pre计算cov3D,cov2d
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,
		geomState.means2D,
		geomState.depths,
		geomState.cov3D,
		geomState.rgb,
		geomState.conic_opacity,
		tile_grid,
		geomState.tiles_touched,
		prefiltered
	), debug)
 /* hipEventRecord(stop);
    hipEventSynchronize(stop); 
	float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("CUDA operation took %f milliseconds\n", milliseconds); */
	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)//排序

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);



    size_t OBB_result_size = num_rendered;
	char* OBB_result_chunkptr = OBBFunc(OBB_result_size);
	bool* OBB_result = reinterpret_cast<bool*>(OBB_result_chunkptr);


    OBB_test << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.point_offsets_old,
		tile_grid,
		radii,
		eigen_lengths,
		eigen_vecs,
		geomState.tiles_touched,
		OBB_result);


	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);


   /*  hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); */
	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets_old,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid,
		OBB_result)
	CHECK_CUDA(, debug)

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);//查找最高比特位





	// Sort complete list of (duplicated) Gaussian indices by keys
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(//对键值进行排序
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)



	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);







	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)//排序
   

 
	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	CHECK_CUDA(FORWARD::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		geomState.means2D,
		feature_ptr,
		geomState.conic_opacity,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		out_color), debug)

	return num_rendered;
	
	 
}




// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(//后向过程中，先render再pre
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	float* dL_dmean2D,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot,
	bool debug)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,//包含瓦片范围的数组
		binningState.point_list,
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor), debug)

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot), debug)
}